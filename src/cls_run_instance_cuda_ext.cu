#include "cls_run_instance_cuda_ext.hpp"

run_instance_cuda_ext::run_instance_cuda_ext() {

}

int run_instance_cuda_ext::getDeviceCount(void) {

  int devices     = 0;

  hipError_t err = hipGetDeviceCount(&devices);

  return devices;

}

run_instance_cuda_ext::~run_instance_cuda_ext() {

}
